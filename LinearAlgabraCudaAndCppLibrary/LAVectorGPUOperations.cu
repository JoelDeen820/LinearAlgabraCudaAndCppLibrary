#include "hip/hip_runtime.h"
/* Author: Joel Deen
* This library takes in arrays of intergers and preforms various vector operations using the GPU.
*/


#include "hip/hip_runtime.h"
#include ""
#include "LAVectorGPUOperations.cuh"
#include <exception>

// GPU kernel Calls ----------------------------------------------------------------------------------------------------------

// just the int add kernal, nothing special for any of the kernals
__global__ void intAdd(int* c, const int* a, const int* b, const unsigned int d)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < d) {
		c[i] = a[i] + b[i];
	}
}

__global__ void intSubtract(int* c, const int* a, const int* b, const unsigned int d) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < d) {
		c[i] = a[i] + b[i];
	}
}

__global__ void intMultiply(int* result, const int* val1, const int val2, const unsigned int size) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		result[blockIdx.x] = val1[blockIdx.x] * val2;
	}
}

__global__ void intDivide(int* result, const int* val1, const int val2, const unsigned int size) { // we are assuming that the user does some prechecking for zero
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < size) {
		result[blockIdx.x] = val1[blockIdx.x] / val2;
	}
}

// CPU Calls -----------------------------------------------------------------------------------------------------------------

int* gpuIntAdd(int* values1, int* values2, unsigned int size) {
	hipError_t cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		throw 1;
	}
	int* result = new int[size];
	int* cuda_vec1;
	int* cuda_vec2;
	int* cuda_result;

	cuda_status = hipMalloc((void**)&cuda_vec1, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}
	cuda_status = hipMalloc((void**)&cuda_vec2, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}
	cuda_status = hipMalloc((void**)&cuda_result, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}

	cuda_status = hipMemcpy(cuda_vec1, values1, size * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		throw 3;
	}
	cuda_status = hipMemcpy(cuda_vec2, values2, size * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		throw 3;
	}
	intAdd << <1 + (size / MAX_ADDING_THREADS), MAX_ADDING_THREADS >> > (cuda_result, cuda_vec1, cuda_vec2, size);

	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		throw 4;
	}

	cuda_status = hipMemcpy(result, cuda_result, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		throw 5;
	}

	cuda_status = hipFree(cuda_vec1);
	if (cuda_status != hipSuccess) {
		throw 6;
	}
	cuda_status = hipFree(cuda_vec2);
	if (cuda_status != hipSuccess) {
		throw 6;
	}
	cuda_status = hipFree(cuda_result);
	if (cuda_status != hipSuccess) {
		throw 6;
	}

	return result;
}

int* gpuIntSubtract(int* values1, int* values2, unsigned int size) {
	hipError_t cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		throw 1;
	}
	int* result = new int[size];
	int* cuda_vec1;
	int* cuda_vec2;
	int* cuda_result;

	cuda_status = hipMalloc((void**)&cuda_vec1, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}
	cuda_status = hipMalloc((void**)&cuda_vec2, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}
	cuda_status = hipMalloc((void**)&cuda_result, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}

	cuda_status = hipMemcpy(cuda_vec1, values1, size * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		throw 3;
	}
	cuda_status = hipMemcpy(cuda_vec2, values2, size * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		throw 3;
	}
	intSubtract << <1 + (size / MAX_ADDING_THREADS), MAX_ADDING_THREADS >> > (cuda_result, cuda_vec1, cuda_vec2, size);
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		throw 4;
	}

	cuda_status = hipMemcpy(result, cuda_result, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		throw 5;
	}

	cuda_status = hipFree(cuda_vec1);
	if (cuda_status != hipSuccess) {
		throw 6;
	}
	cuda_status = hipFree(cuda_vec2);
	if (cuda_status != hipSuccess) {
		throw 6;
	}
	cuda_status = hipFree(cuda_result);
	if (cuda_status != hipSuccess) {
		throw 6;
	}

	return result;
}

int* gpuIntMultiply(int* values1, int value, unsigned int size) {
	hipError_t cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		throw 1;
	}
	int* result = new int[size];
	int* cuda_vec1;
	int* cuda_result;

	cuda_status = hipMalloc((void**)&cuda_vec1, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}
	cuda_status = hipMalloc((void**)&cuda_result, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}

	cuda_status = hipMemcpy(cuda_vec1, values1, size * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		throw 3;
	}
	intMultiply << <1 + (size / MAX_ADDING_THREADS), MAX_ADDING_THREADS >> > (cuda_result, cuda_vec1, value, size);
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		throw 4;
	}

	cuda_status = hipMemcpy(result, cuda_result, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		throw 5;
	}

	cuda_status = hipFree(cuda_vec1);
	if (cuda_status != hipSuccess) {
		throw 6;
	}
	cuda_status = hipFree(cuda_result);
	if (cuda_status != hipSuccess) {
		throw 6;
	}

	return result;
}

int* gpuIntDivide(int* values1, int value, unsigned int size) {
	if (value == 0) {
		throw 7;
	}
	hipError_t cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		throw 1;
	}
	int* result = new int[size];
	int* cuda_vec1;
	int* cuda_result;

	cuda_status = hipMalloc((void**)&cuda_vec1, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}
	cuda_status = hipMalloc((void**)&cuda_result, size * sizeof(int));
	if (cuda_status != hipSuccess) {
		throw 2;
	}

	cuda_status = hipMemcpy(cuda_vec1, values1, size * sizeof(int), hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		throw 3;
	}
	intMultiply << <1 + (size / MAX_ADDING_THREADS), MAX_ADDING_THREADS >> > (cuda_result, cuda_vec1, value, size);
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		throw 4;
	}

	cuda_status = hipMemcpy(result, cuda_result, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		throw 5;
	}

	cuda_status = hipFree(cuda_vec1);
	if (cuda_status != hipSuccess) {
		throw 6;
	}
	cuda_status = hipFree(cuda_result);
	if (cuda_status != hipSuccess) {
		throw 6;
	}

	return result;
}